
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

const size_t size = 1 << 20;

__global__ void transpose(float_t *matrixOrigin, float_t *matrixRes) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t width = gridDim.x * blockDim.x;
    matrixRes[x + y * width] = matrixOrigin[y + x * width];
}

__global__ void saxpy(float_t *vectorA, float_t *vectorB, float_t alpha) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    vectorA[index] = vectorA[index] * alpha + vectorB[index];
}

int32_t main() {
    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
    hipStream_t stream0;
    const size_t num = 32;
    const size_t Nx = 1 << 10;
    const size_t Ny = 1 << 10;
    hipStreamCreate(&stream0);
    float_t *matrix, *matrix_dev_origin, *matrix_dev_res;
    hipHostAlloc((void **) &matrix, size * sizeof(float_t), hipHostMallocDefault);
    for (int64_t i = 0; i < size; ++i)
        matrix[i] = i;
    hipMalloc((void **) &matrix_dev_origin, sizeof(float_t) * size);
    hipMalloc((void **) &matrix_dev_res, sizeof(float_t) * size);

    hipMemcpyAsync(matrix_dev_origin, matrix, sizeof(float_t) * size, hipMemcpyHostToDevice, stream0);
    transpose <<< dim3(Nx / num, Ny / num), dim3(num, num) >>>(matrix_dev_origin, matrix_dev_res);
    hipMemcpyAsync(matrix, matrix_dev_res, sizeof(float_t) * size, hipMemcpyDeviceToHost, stream0);
    hipStreamSynchronize(stream0);
    std::chrono::high_resolution_clock::time_point stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double_t> time_span = std::chrono::duration_cast<std::chrono::duration<double_t>>(
            stop - start);
    std::cout << "Transpose time (s) - " << time_span.count() << std::endl;
//        for(int64_t i = 0; i < Ny; ++i)
//        {
//            for(int64_t j = 0; j < Nx; ++j)
//                std:: cout << matrix[i * Nx + j] << " ";
//            std::cout << std::endl;
//        }
    hipFree(matrix_dev_origin);
    hipFree(matrix_dev_res);
    hipHostFree(matrix);
    start = std::chrono::high_resolution_clock::now();
    float_t *vecA, *vecB, *vecA_device, *vecB_device;
    hipStream_t stream_m0;
    hipStreamCreate(&stream_m0);
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipHostAlloc((void **) &vecA, size * sizeof(float_t), hipHostMallocDefault);
    hipHostAlloc((void **) &vecB, size * sizeof(float_t), hipHostMallocDefault);
    for (int64_t i = 0; i < size; ++i) {
        vecA[i] = i;
        vecB[i] = i * 2 - 1;
    }
    hipMalloc((void **) &vecA_device, sizeof(float_t) * size);
    hipMalloc((void **) &vecB_device, sizeof(float_t) * size);
    hipMemcpyAsync(vecA_device, vecA, sizeof(int) * size, hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(vecB_device, vecB, sizeof(int) * size, hipMemcpyHostToDevice, stream1);
    saxpy <<< size / 2 / 1024, 1024, 0, stream0 >>>(vecA_device, vecB_device, 2.25);
    saxpy <<< size / 2 / 1024, 1024, 0, stream1 >>>(vecA_device + size / 2, vecB_device + size / 2, 2.25);
    hipMemcpyAsync(vecA, vecA_device, sizeof(float_t) * size / 2, hipMemcpyDeviceToDevice, stream0);
    hipMemcpyAsync(vecA + size / 2, vecA_device + size / 2, sizeof(float_t) * size / 2, hipMemcpyDeviceToDevice,
                    stream1);
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    stop = std::chrono::high_resolution_clock::now();
    time_span = std::chrono::duration_cast<std::chrono::duration<double_t>>(stop - start);
    std::cout << "SAXPY time (s) - " << time_span.count() << std::endl;
    //for (int64_t i = 0; i < size; ++i)
    //	std::cout << vecA[i] << " ";
    hipFree(vecA_device);
    hipFree(vecB_device);
    hipHostFree(vecA);
    hipHostFree(vecB);
}