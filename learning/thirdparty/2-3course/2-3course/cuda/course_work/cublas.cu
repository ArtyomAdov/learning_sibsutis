
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <cstddef>
#include <iomanip>
#include <chrono>


int main()
{
    size_t cols = 1 << 10;
    size_t rows = 1 << 10;
    size_t N = cols * rows;
    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float_t* matrix;
    hipHostMalloc((void**)&matrix, N * sizeof(float_t), hipHostMallocDefault);
    for (int i = 0; i < N; ++i)
        matrix[i] = static_cast<float_t>(i);
    float_t* matrix_in_dev;
    hipMalloc((void**)&matrix_in_dev, N * sizeof(float_t));
    float_t* matrix_out_dev;
    hipMalloc((void**)&matrix_out_dev, N * sizeof(float_t));
    hipblasSetMatrix(rows, cols, sizeof(float_t), matrix, rows, matrix_in_dev, rows);
    float_t alpha = 1.; // change sample
    float_t beta = 0.;
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, cols, rows, &alpha, matrix_in_dev, rows, &beta, matrix_in_dev, rows, matrix_out_dev, cols);
    hipblasGetMatrix(rows, cols, sizeof(float_t), matrix_out_dev, rows, matrix, rows);
    hipStreamSynchronize(nullptr);
    std::chrono::high_resolution_clock::time_point stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double_t> time_span = std::chrono::duration_cast<std::chrono::duration<double_t>>(stop - start);
//       for (int i = 0; i < rows; ++i)
//        {
//            for (int j = 0; j < cols; ++j)
//                std::cout << std::setw(10) << static_cast<float>(matrix[j + i * rows]) << "\t";
//            std::cout << "\n";
//        }
    hipHostFree(matrix);
    hipFree(matrix_in_dev);
    hipFree(matrix_out_dev);
    hipblasDestroy(handle);
    std::cout << "Matrix transpose (s) " << "  -  " << time_span.count() << "\n";
    hipblasHandle_t handle1;
    hipblasCreate(&handle1);

    start = std::chrono::high_resolution_clock::now();
    float_t* vecA;
    hipHostMalloc((void**)&vecA, N * sizeof(float_t), hipHostMallocDefault);
    float_t* vecB;
    hipHostMalloc((void**)&vecB, N * sizeof(float_t), hipHostMallocDefault);

    for (int i = 0; i < N; ++i)
    {
        vecA[i] = (float_t)i;
        vecB[i] = (float_t)(i * 2 - 1);
    }
    float_t* aDev;
    hipMalloc((void**)&aDev, N * sizeof(float_t));
    float_t* bDev;
    hipMalloc((void**)&bDev, N * sizeof(float_t));

    hipblasSetMatrix(N, 1, sizeof(float_t), vecA, N, aDev, N);
    hipblasSetMatrix(N, 1, sizeof(float_t), vecB, N, bDev, N);
    alpha = 2.25;
    hipblasSaxpy(handle1, N, &alpha, aDev, 1, bDev, 1);
    hipblasGetMatrix(N, 1, sizeof(float_t), bDev, N, vecB, N);
    hipStreamSynchronize(nullptr);
    //for (int i = 0; i < N; ++i)
    //	printf("%f\n", vecB[i]);
    hipblasDestroy(handle1);
    hipHostFree(vecA);
    hipHostFree(vecB);
    hipFree(aDev);
    hipFree(bDev);
    stop = std::chrono::high_resolution_clock::now();
    time_span = std::chrono::duration_cast<std::chrono::duration<double_t>>(stop - start);
    std::cout << "Saxpy time (s) " << "  -  " << time_span.count() << "\n";
}